#include "hip/hip_runtime.h"


#include "stdafx.h"
#ifdef IS_GPU
#include "snapshot.h"
#include "make_simulation.h" //version information is stored at make_simulation.h
#define numOfThrPerBlock 512 //maximum value of 512 for cuda compute capabiltiy 1.3 or below, 1024 for 2.0 or above

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>



//defines the variables used on the device
class GPU_vars
{
public:
	GPU_vars()
	{
		kernel = NULL;
		tau_l = NULL;
		tau_n = NULL;
		tau_p = NULL;
		pos = NULL;
	}

	deftype * kernel;
	deftype * tau_l;
	deftype * tau_n;
	deftype * tau_p;
	int * pos;
	place * minPoint;
	deftype * deform;
};


//allocate the required memory on the device and copy the required variables to the device
bool allocateAndCopyToDevice(simVars& sim, GPU_vars& d)
{
	int size = sim.getSize();
	int linsize = size * size;

	hipError_t cudaStatus;
	if ((cudaStatus = hipMalloc(&d.kernel,sizeof(deftype) * linsize)) != hipSuccess ||
		(cudaStatus = hipMalloc(&d.tau_l,sizeof(deftype) * linsize)) != hipSuccess ||
		(cudaStatus = hipMalloc(&d.tau_n,sizeof(deftype) * linsize)) != hipSuccess ||
		(cudaStatus = hipMalloc(&d.tau_p,sizeof(deftype) * linsize)) != hipSuccess ||
		(cudaStatus = hipMalloc(&d.pos,sizeof(int) * linsize)) != hipSuccess ||
		(cudaStatus = hipMalloc(&d.minPoint,sizeof(place))) != hipSuccess ||
		(cudaStatus = hipMalloc(&d.deform,sizeof(deftype))) != hipSuccess)
	{
		cerr << "hipMalloc failed, " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	if ((cudaStatus = hipMemcpy(d.kernel,&sim.sf[0],sizeof(deftype) * linsize,hipMemcpyHostToDevice)) != hipSuccess ||
		(cudaStatus = hipMemcpy(d.tau_l,&sim.tau_l[0],sizeof(deftype) * linsize,hipMemcpyHostToDevice)) != hipSuccess ||
		(cudaStatus = hipMemcpy(d.tau_n,&sim.tau_n[0],sizeof(deftype) * linsize,hipMemcpyHostToDevice)) != hipSuccess ||
		(cudaStatus = hipMemcpy(d.tau_p,&sim.tau_p[0],sizeof(deftype) * linsize,hipMemcpyHostToDevice)) != hipSuccess)
	{
		cerr << "hipMemcpyHostToDevice failed, " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	return true;
}


//copy back the tau_l space
//tau_l cannot be in a synchronised state with host device in an effective way
bool tau_lCopyToHost(simVars& sim, GPU_vars& d)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(&sim.tau_l[0],d.tau_l,sizeof(deftype) * sim.getSize() * sim.getSize(),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cerr << "hipMemcpy(&sim.tau_l,d.tau_l,sizeof(int) * sim.getSize() * sim.getSize(),hipMemcpyDeviceToHost) failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	return true;
}


//copy back the tau_n and tau_p space
//these variables can be in a synchronised state with the host device
bool tau_npCopytoHost(simVars& sim, GPU_vars& d)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(&sim.tau_n[0],d.tau_n,sizeof(deftype) * sim.getSize() * sim.getSize(),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		cerr << "hipMemcpy(&sim.tau_n,d.tau_n,sizeof(int) * sim.getSize() * sim.getSize(),hipMemcpyDeviceToHost) failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipMemcpy(&sim.tau_p[0],d.tau_p,sizeof(deftype) * sim.getSize() * sim.getSize(),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		cerr << "hipMemcpy(&sim.tau_p,d.tau_p,sizeof(int) * sim.getSize() * sim.getSize(),hipMemcpyDeviceToHost) failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	return true;
}

__global__ void resetPos(int * pos)
{
	const int id = threadIdx.x + blockIdx.x * blockDim.x;
	pos[id] = id;
}

/// <summary>Find minimum values in array of size of threadPerblock</summary>
/// <param name="linsize">The number of elemets that has to compared.
/// It is not sure that every elemets will be compared to every otther one,
/// but minimum values will be selected from every threadPerblock number of elements.</param>
/// <param name="threadPower">The lb of the minimum distance - 1 between two cells that has to compared. It is equal with lb(sim.getSize() * sim.getSize()  / linsize) - 1.</param>
__global__ void findMinIterate(int threadPower, deftype tau_ext, deftype * tau_l, deftype * tau_n, deftype * tau_p, int * pos)
{
	const int id = (threadIdx.x + (blockIdx.x * blockDim.x << 1)) << threadPower;

	//find the minimum place
	for (int i = blockDim.x; i > 0; i>>=1)
	{
		if (threadIdx.x < i)
		{
			deftype tau_act_A, tau_act_A_l, tau_act_B, tau_act_B_l;
			
			tau_act_A   = tau_p[pos[id]] - tau_ext - tau_l[pos[id]];
			tau_act_A_l = tau_n[pos[id]] + tau_ext + tau_l[pos[id]];
			if (tau_act_A_l < 0 && tau_act_A_l < tau_act_A)
				tau_act_A = tau_act_A_l;

			tau_act_B   = tau_p[pos[id + (i<<threadPower)]] - tau_ext - tau_l[pos[id + (i<<threadPower)]];
			tau_act_B_l = tau_n[pos[id + (i<<threadPower)]] + tau_ext + tau_l[pos[id + (i<<threadPower)]];
			if (tau_act_B_l < 0 && tau_act_B_l < tau_act_B)
				tau_act_B = tau_act_B_l;

			if (tau_act_B < tau_act_A)
				pos[id] = pos[id + (i<<threadPower)];

		}
		__syncthreads();
	}
}

__global__ void setMinPointResetPos(int size, int sizePower, deftype tau_ext, deftype * tau_l, deftype * tau_n, deftype * tau_p, int * pos, place * minPoint)
{
	const int id = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (id == 0)
	{
		minPoint->tau_l = tau_l[pos[0]];
		minPoint->x = pos[0] >> sizePower;
		minPoint->y = pos[0] & (size-1);
	}
	pos[id] = id;
}

__global__ void addKernelRefreshYield(int size, int power, deftype * kernel, deftype * tau_l, deftype * tau_n, deftype * tau_p, place * minPoint, direction dir, deftype newYield, deftype deform)
{
	const int id = threadIdx.x + blockIdx.x * blockDim.x;
	
	//x and y values are needed to correctly referr to kernel values
	const int prevX = minPoint->x;
	const int prevY = minPoint->y;
	int prevPos = (prevX << power) + prevY;

	const int idx = id >> power;
	const int idy = id & (size-1);

	//refresh the Yield point at the previous deformation
	if (id == prevPos)
	{
		if (dir == direction::left)
			tau_n[prevPos] = newYield;
		else
			tau_p[prevPos] = newYield;
	}

	tau_l[id] += kernel[(((idx-prevX+size) & (size - 1)) << power) + ((idy-prevY+size) & (size - 1))] * deform;
}




//find the minimum points, where deformation incidence will appear
//and also its direction and value
bool findMin(simVars& sim, place& minPoint, GPU_vars& d)
{
	resetPos<<<sim.getSize() * sim.getSize() / numOfThrPerBlock,numOfThrPerBlock>>>(d.pos);
	hipError_t cudaStatus;
	
#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "resetPos launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching resetPos!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion
	int threadPower = 0;
	const int iterationPower = getPower(numOfThrPerBlock) + 1;
	const int linSize = sim.getSize() * sim.getSize();
	for (int restSize =  linSize; restSize > 1; restSize /= numOfThrPerBlock * 2)
	{
		int numOfBlock = restSize / (2 * numOfThrPerBlock);
		if (numOfBlock == 1 || numOfBlock == 0)
		{
			findMinIterate<<<1,restSize/2>>>(threadPower, sim.tau_ext, d.tau_l, d.tau_n, d.tau_p, d.pos);
#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "findMinIterate<<<1,numOfThrPerBlock>>> launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching findMinIterate<<<1,numOfThrPerBlock>>>!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion
			break;
		}
		findMinIterate<<<numOfBlock,numOfThrPerBlock>>>(threadPower, sim.tau_ext, d.tau_l, d.tau_n, d.tau_p, d.pos);
#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "findMinIterate<<<numOfBlock,numOfThrPerBlock>>> launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching findMinIterate<<<numOfBlock,numOfThrPerBlock>>>!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion
		threadPower += iterationPower;
	}

	setMinPointResetPos<<<sim.getSize() * sim.getSize() / numOfThrPerBlock,numOfThrPerBlock>>>(sim.getSize(), sim.getPower(), sim.tau_ext, d.tau_l, d.tau_n, d.tau_p, d.pos, d.minPoint);

#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "setMinPointResetPos launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching setMinPointResetPos!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion

	cudaStatus = hipMemcpy(&minPoint,d.minPoint,sizeof(place),hipMemcpyDeviceToHost);

#pragma region check cudaStatus
#ifdef _DEBUG
	if (cudaStatus != hipSuccess)
	{
		cerr << "hipMemcpy(&minPoint,d.minPoint,sizeof(place),hipMemcpyDeviceToHost) failed, " << hipGetErrorString(cudaStatus) << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion

	return true;
}


//at minPoint, calclulates the needed deformation based on tau_l and
//             refresh the tau_l field with the previous deformation placed at minPoint
//than finds the minimum point and
//refresh the yield point of the deformed cell
bool refreshFindMin(const simVars& sim, place& minPoint, direction nowFlow, deftype deform, GPU_vars& d)
{
	hipError_t cudaStatus;
	
	deftype newYield = (nowFlow == direction::right) ? sim.tau_p(minPoint.x,minPoint.y) : sim.tau_n(minPoint.x,minPoint.y);
	addKernelRefreshYield<<<sim.getSize() * sim.getSize() / numOfThrPerBlock,numOfThrPerBlock>>>(
		sim.getSize(),
		sim.getPower(),
		d.kernel, d.tau_l, d.tau_n, d.tau_p, d.minPoint,
		nowFlow,
		newYield,
		deform);

#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "addKernelRefreshYield launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching addKernelRefreshYield!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion

	int threadPower = 0;
	const int iterationPower = getPower(numOfThrPerBlock) + 1;
	const int linSize = sim.getSize() * sim.getSize();
	for (int restSize =  linSize; restSize > 1; restSize /= numOfThrPerBlock * 2)
	{
		int numOfBlock = restSize / (2 * numOfThrPerBlock);
		if (numOfBlock == 1 || numOfBlock == 0)
		{
			findMinIterate<<<1,restSize/2>>>(threadPower, sim.tau_ext, d.tau_l, d.tau_n, d.tau_p, d.pos);
#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "findMinIterate<<<1,numOfThrPerBlock>>> launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching findMinIterate<<<1,numOfThrPerBlock>>>!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion
			break;
		}
		findMinIterate<<<numOfBlock,numOfThrPerBlock>>>(threadPower, sim.tau_ext, d.tau_l, d.tau_n, d.tau_p, d.pos);
#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "findMinIterate<<<numOfBlock,numOfThrPerBlock>>> launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching findMinIterate<<<numOfBlock,numOfThrPerBlock>>>!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion
		threadPower += iterationPower;
	}

	setMinPointResetPos<<<sim.getSize() * sim.getSize() / numOfThrPerBlock,numOfThrPerBlock>>>(sim.getSize(), sim.getPower(), sim.tau_ext, d.tau_l, d.tau_n, d.tau_p, d.pos, d.minPoint);

#pragma region check cudaStatus
#ifdef _DEBUG
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cerr << "setMinPointResetPos launch failed: " << hipGetErrorString(cudaStatus) << endl;;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cerr << "hipDeviceSynchronize returned error code " << hipGetErrorString(cudaStatus) << " after launching setMinPointResetPos!" << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion

	cudaStatus = hipMemcpy(&minPoint,d.minPoint,sizeof(place),hipMemcpyDeviceToHost);

#pragma region check cudaStatus
#ifdef _DEBUG
	if (cudaStatus != hipSuccess)
	{
		cerr << "hipMemcpy(&minPoint,d.minPoint,sizeof(place),hipMemcpyDeviceToHost) failed, " << hipGetErrorString(cudaStatus) << endl;
		hipFree(d.kernel);
		hipFree(d.tau_l);
		hipFree(d.tau_n);
		hipFree(d.tau_p);
		hipFree(d.pos);
		hipFree(d.minPoint);
		hipFree(d.deform);
		return false;
	}
#endif
#pragma endregion
	
	return true;

}


bool simVars::makeDeformSerial_GPU(const simPars& pars, int nominalDef, bool printToTau_g)
{

	time_t lastWrite = time(NULL); 

	place minPoint;

	GPU_vars d;

#pragma region device_setup

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus;
    if (hipSetDevice(0) != hipSuccess) {
        cerr << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << endl;;
        return false;
    }
	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    if (hipDeviceReset() != hipSuccess) {
        cerr << "hipDeviceReset failed!" << endl;;
        return false;
    }
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		cerr << "hipDeviceSynchronize failed, " << hipGetErrorString(cudaStatus) << endl;
		return false;
	}
	if (hipDeviceSynchronize() != hipSuccess)
	{
		cerr << "hipDeviceSynchronize failed" << endl;
		return false;
	}
	
	if (!allocateAndCopyToDevice(*this,d))
	{
		cerr << "Error at allocateAndCopyToDevice" << endl;
		return false;
	}
#pragma endregion
	
	if (!findMin(*this,minPoint,d))
	{
		cerr << "Error at findMin" << endl;
		return false;
	}

	printInfDescr(pars);

	ofstream tau_g(fnameWp("tau_g",pars),ios_base::app);
	if (!tau_g)
	{
		cerr << "Cannot app to " << fnameWp("tau_g",pars) << endl;
		return false;
	}
	tau_g.precision(17); //http://en.wikipedia.org/wiki/Double-precision_floating-point_format#IEEE_754_double-precision_binary_floating-point_format:_binary64
	while (getDeform() < nominalDef)
	{
		if (DEBUG_TEST)
			printInf(minPoint,*this,pars);

		if (willYield(*this,minPoint) == none) //el�fordulhatna am�gy, h a bet�lt�skor �pp egy lavina van, �s akkor nem kell megn�velni; teh�t ha egy lavina elej�n vagyunk
		{
			tau_ext =  tau_p(minPoint.x,minPoint.y) - minPoint.tau_l + EPSILON;
			av_size = 0;
			av_size_n = 0;
		}

		do
		{
			if (static_cast<int>(time(NULL) - lastWrite) > pars.tbs()) //Gamma and tau_l are in consistent state, not as like at the end of this loop
			{
				tau_lCopyToHost(*this,d); //copy back the data from the device
				tau_npCopytoHost(*this,d);
				snapshot bak(*this,static_cast<int>(getDeform()));
				bak.make(pars);
				delMarkedSnapshots(pars);
				bak.mark(pars);
				lastWrite = time_t(0);
			}
			
			direction nowFlow = willYield(*this,minPoint);
			deftype deform = calcDeform(minPoint,tau_ext,sf[0],pars.DG(),nowFlow);
			Gamma[minPoint.x * size + minPoint.y] += deform;
			if (nowFlow == direction::right)
			{
				av_size += deform;
				sG += deform;
			}
			else // nowFlow == direction::left must hold
			{
				av_size_n -= deform; //av_size_n stay positive
				sGn -= deform; //sGn stay positive
			}

			if (pars.fsrg())
			{
				if (nowFlow == direction::right)
				{
					tau_p(minPoint.x, minPoint.y) = rand_gen();
					if (pars.lfs().MyType == leftFlowStress::same)
						tau_n(minPoint.x, minPoint.y) = tau_p(minPoint.x, minPoint.y);
				}
				else // nowFlow == direction::left
				{
					tau_n(minPoint.x, minPoint.y) = rand_gen();
					if (pars.lfs().MyType == leftFlowStress::same)
						tau_p(minPoint.x, minPoint.y) = tau_n(minPoint.x, minPoint.y);
				}
			}
				
			if (!refreshFindMin(*this,minPoint,nowFlow,deform,d))
			{
				cerr << "Error at refreshFindMin" << endl;
				return false;
			}

			if (DEBUG_TEST)
				printInf(minPoint,*this,pars);

		} while (willYield(*this,minPoint) != direction::none && getDeform() < nominalDef);

		if (willYield(*this,minPoint) == direction::none && printToTau_g)
			printTau_g(tau_g);
	}
	
	tau_lCopyToHost(*this,d);
	tau_npCopytoHost(*this,d);
	if (!snapshot(*this,static_cast<int>(getDeform())).make(pars) || !delMarkedSnapshots(pars))
		return false;
	
	hipFree(d.kernel);
	hipFree(d.tau_l);
	hipFree(d.tau_n);
	hipFree(d.tau_p);
	hipFree(d.pos);
	hipFree(d.minPoint);
	hipFree(d.deform);

	return true;
}

#endif